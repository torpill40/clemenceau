
#include <hip/hip_runtime.h>
extern "C" __global__ void gen_binary_map(int *map, int m, int n)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m && j < n)
    {
        map[i * n + j] = (i >> j) & 1;
    }
}
